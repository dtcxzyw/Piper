#include "hip/hip_runtime.h"
#pragma once
#define __int32 int
#define __int64 long long
#define _NODISCARD
#define WCHAR_MIN 0x0000
#define WCHAR_MAX 0xffff
#define CUDA_VERSION 10020
namespace std {
    typedef signed char int8_t;
    typedef short int16_t;
    typedef int int32_t;
    typedef long long int64_t;
    typedef unsigned char uint8_t;
    typedef unsigned short uint16_t;
    typedef unsigned int uint32_t;
    typedef unsigned long long uint64_t;
    typedef uint64_t size_t;
    template <typename genType>
    struct make_unsigned {};

    template <>
    struct make_unsigned<char> {
        typedef unsigned char type;
    };

    template <>
    struct make_unsigned<signed char> {
        typedef unsigned char type;
    };

    template <>
    struct make_unsigned<short> {
        typedef unsigned short type;
    };

    template <>
    struct make_unsigned<int> {
        typedef unsigned int type;
    };

    template <>
    struct make_unsigned<long> {
        typedef unsigned long type;
    };

    template <>
    struct make_unsigned<int64_t> {
        typedef uint64_t type;
    };

    template <>
    struct make_unsigned<unsigned char> {
        typedef unsigned char type;
    };

    template <>
    struct make_unsigned<unsigned short> {
        typedef unsigned short type;
    };

    template <>
    struct make_unsigned<unsigned int> {
        typedef unsigned int type;
    };

    template <>
    struct make_unsigned<unsigned long> {
        typedef unsigned long type;
    };

    template <>
    struct make_unsigned<uint64_t> {
        typedef uint64_t type;
    };

    using ::acos;
    using ::asin;
    using ::atan;
    using ::ceil;
    using ::cos;
    using ::cosh;
    using ::exp;
    using ::floor;
    using ::log;
    using ::pow;
    using ::sin;
    using ::sinh;
    using ::sqrt;
    using ::tan;
    using ::tanh;
}  // namespace std
constexpr long long operator"" i64(unsigned long long x) {
    return x;
}
constexpr unsigned long long operator"" ui64(unsigned long long x) {
    return x;
}
using std::uint32_t;
using std::uint64_t;
#define _HUGE_ENUF 1e+300  // _HUGE_ENUF*_HUGE_ENUF must overflow
#define INFINITY ((float)(_HUGE_ENUF * _HUGE_ENUF))
#define HUGE_VAL ((double)INFINITY)
#define HUGE_VALF ((float)INFINITY)
#define __builtin_huge_val()  HUGE_VAL
#define __builtin_huge_valf()  HUGE_VALF 
#define __builtin_nan nan
#define __builtin_nanf  nanf 
#define __builtin_nans nan
#define __builtin_nansf nanf
#include "../Shared/KernelShared.hpp"
