#include "hip/hip_runtime.h"
#include "../../Shared/KernelShared.hpp"
#include "DataDesc.hpp"

GLOBAL void __raygen__renderKernel() {
    auto data = getSBTData<DataDesc>();
    uint3 pixelPos = optixGetLaunchIndex();
    Spectrum acc = {};
    unsigned count = 0;
    for(unsigned id = data->sampleIdxBeg; id < data->sampleIdxEnd; ++id) {
        SamplerInitResult initRes = initSampler(id, pixelPos.x, pixelPos.y);
        SamplerContext sampler;
        sampler.dim = 0, sampler.index = initRes.index;
        RaySample ray =
            generateRay(data->generateRay, initRes.px, initRes.py, sampler);
        Spectrum res = sampleOnePixel(data->sampleOnePixel, ray, &sampler);
        if(data->filtBadColor &
           !(isfinite(res.x) & isfinite(res.y) & isfinite(res.z)))
            continue;
        acc += res;
        ++count;
    }
    data->outputBuffer[data->width * pixelPos.y + pixelPos.x] +=
        Vec4(acc, static_cast<float>(count));
}

GLOBAL void __miss__rad() {}
GLOBAL void __miss__occ() {
    optixSetPayload_0(1);
}
GLOBAL void __exception__default() {
    optix_impl::optixDumpExceptionDetails();
}
GLOBAL void __exception__silence() {}
