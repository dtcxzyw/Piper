#include "hip/hip_runtime.h"
#include "../../Shared/KernelShared.hpp"
#include "DataDesc.hpp"

DEVICE void __raygen__renderKernel() {
    auto data = getSBTData<DataDesc>();
    uint3 pixelPos = optixGetLaunchIndex();
    Spectrum acc = {};
    unsigned count = 0;
    for(unsigned id = data->sampleIdxBeg; id < data->sampleIdxEnd; ++id) {
        SamplerInitResult initRes = initSampler(id, pixelPos.x, pixelPos.y);
        SamplerContext sampler;
        sampler.dim = 0, sampler.index = initRes.index;
        RaySample ray =
            generateRay(data->generateRay, initRes.px, initRes.py, sampler);
        Spectrum res = sampleOnePixel(data->sampleOnePixel, ray, &sampler);
        if(data->filtBadColor &
           !(isfinite(res.x) & isfinite(res.y) & isfinite(res.z)))
            continue;
        acc += res;
        ++count;
    }
    data->outputBuffer[data->width * pixelPos.y + pixelPos.x] +=
        Vec4(acc, static_cast<float>(count));
}

DEVICE void __miss__occ() {
    optixSetPayload_0(1);
}
DEVICE void __exception__default() {
    optix_impl::optixDumpExceptionDetails();
}
DEVICE void __exception__silence() {}
