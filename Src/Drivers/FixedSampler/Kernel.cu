#include "hip/hip_runtime.h"
#include "../../Shared/KernelShared.hpp"
#include "DataDesc.hpp"

GLOBAL void __raygen__renderKernel() {
    const DataDesc* data =
        reinterpret_cast<DataDesc*>(optixGetSbtDataPointer());
    uint3 pixelPos = optixGetLaunchIndex();
    unsigned id = data->width * (data->sampleIdx * data->height + pixelPos.y) +
        pixelPos.x;
    Vec4 res = optixContinuationCall<Vec4, unsigned, Uint2>(
        static_cast<unsigned>(SBTSlot::samplePixel), id,
        Uint2{ pixelPos.x, pixelPos.y });
    if(data->filtBadColor &
       !(isfinite(res.x) & isfinite(res.y) & isfinite(res.z) & isfinite(res.w)))
        return;
    data->outputBuffer[data->width * pixelPos.y + pixelPos.x] += res;
}

GLOBAL void __miss__rad() {}
GLOBAL void __miss__occ() {
    optixSetPayload_0(1);
}
GLOBAL void __exception__empty() {}
