#include "hip/hip_runtime.h"
#include "../../Shared/KernelShared.hpp"
#include "DataDesc.hpp"

GLOBAL void __raygen__renderKernel() {
    const DriverData* data =
        reinterpret_cast<DriverData*>(optixGetSbtDataPointer());
    uint3 pixelPos = optixGetLaunchIndex();
    unsigned id = data->width * (data->sampleIdx * data->height + pixelPos.y) +
        pixelPos.x;
    Vec4 res = optixDirectCall<Vec4, unsigned, Uint2>(
        static_cast<unsigned>(SBTSlot::samplePixel), id,
        Uint2{ pixelPos.x, pixelPos.y });
    if(data->filtBadColor &
       !(isfinite(res.x) & isfinite(res.y) & isfinite(res.z) & isfinite(res.w)))
        return;
    data->outputBuffer[data->width * pixelPos.y + pixelPos.x] += res;
}

GLOBAL void __miss__rad() {}
GLOBAL void __miss__occ() {}
GLOBAL void __exception__empty() {}
