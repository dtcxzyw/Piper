#include "hip/hip_runtime.h"
#include "../../Shared/KernelShared.hpp"
#include "DataDesc.hpp"

GLOBAL void __raygen__renderKernel() {
    auto data = getSBTData<DataDesc>();
    uint3 pixelPos = optixGetLaunchIndex();
    Vec4 res = samplePixel(data->sampleIdx, pixelPos.x, pixelPos.y);
    if(data->filtBadColor &
       !(isfinite(res.x) & isfinite(res.y) & isfinite(res.z) & isfinite(res.w)))
        return;
    data->outputBuffer[data->width * pixelPos.y + pixelPos.x] += res;
}

GLOBAL void __miss__rad() {}
GLOBAL void __miss__occ() {
    optixSetPayload_0(1);
}
GLOBAL void __exception__default() {
    int code = optixGetExceptionCode();
    if(code == OPTIX_EXCEPTION_CODE_TRAVERSAL_INVALID_HIT_SBT)
        printf("Invalid hit sbt idx=%d,off=%d\n",
               optixGetExceptionInvalidSbtOffset(), optixGetPrimitiveIndex());
    else
        printf("exception %d\n", code);
}
GLOBAL void __exception__silence() {}
