#include "hip/hip_runtime.h"
#include "../../Shared/KernelShared.hpp"
#include "DataDesc.hpp"

GLOBAL void __closesthit__RCH() {
    printf("hit\n");
    const DataDesc* data =
        reinterpret_cast<DataDesc*>(optixGetSbtDataPointer());
    float2 vw = optixGetTriangleBarycentrics();
    Payload* payload = getPayload();
    payload->hit = false;
    payload->rad =
        Spectrum{ 1.0f, 1.0f,
                  1.0f };  // Spectrum{ 1.0f - vw.x - vw.y, vw.x, vw.y };
}

GLOBAL void __anyhit__OAH() {}
