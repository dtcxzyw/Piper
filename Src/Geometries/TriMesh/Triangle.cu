#include "hip/hip_runtime.h"
#include "../../Shared/KernelShared.hpp"
#include "DataDesc.hpp"

GLOBAL void __closesthit__RCH() {
    const DataDesc* data = getSBTData<DataDesc>();
    float2 uv = optixGetTriangleBarycentrics();
    float u = uv.x, v = uv.y, w = 1.0f - u - v;
    Payload* payload = getPayload();
    Uint3 idx = data->index[optixGetPrimitiveIndex()];
    Vec3 p0 = data->vertex[0], p1 = data->vertex[1], p2 = data->vertex[2];
    Vec3 ng = glm::cross(p1 - p0, p2 - p0);
    bool front = optixIsTriangleFrontFaceHit();
    ng = (front ? ng : -ng);
    Vec3 ns;
    if(data->normal) {
        ns = data->normal[idx.x] * u + data->normal[idx.y] * v +
            data->normal[idx.z] * w;
        ns = (front ? ns : -ns);
    } else
        ns = ng;

    Vec2 texCoord = { 0.0f, 0.0f };
    if(data->texCoord)
        texCoord = data->texCoord[idx.x] * u + data->texCoord[idx.y] * v +
            data->texCoord[idx.z] * w;
    payload->hit = false;
    payload->rad = glm::normalize(glm::abs(ns));
}

GLOBAL void __anyhit__OAH() {}
