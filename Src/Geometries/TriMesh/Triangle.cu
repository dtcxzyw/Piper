#include "hip/hip_runtime.h"
#include "../../Shared/KernelShared.hpp"
#include "DataDesc.hpp"

GLOBAL void __closesthit__RCH() {
    const DataDesc* data = getSBTData<DataDesc>();
    float2 uv = optixGetTriangleBarycentrics();
    float u = uv.x, v = uv.y, w = 1.0f - u - v;
    Payload* payload = getPayload();
    Uint3 idx = data->index[optixGetPrimitiveIndex()];
    Vec3 p0 = f2v(
             optixTransformPointFromObjectToWorldSpace(v2f(data->vertex[0]))),
         p1 = f2v(
             optixTransformPointFromObjectToWorldSpace(v2f(data->vertex[1]))),
         p2 = f2v(
             optixTransformPointFromObjectToWorldSpace(v2f(data->vertex[2])));
    Vec3 ng = glm::cross(p1 - p0, p2 - p0);
    bool front = optixIsTriangleFrontFaceHit();
    Vec3 ns;
    if(data->normal) {
        ns = data->normal[idx.x] * u + data->normal[idx.y] * v +
            data->normal[idx.z] * w;
        ns = f2v(optixTransformNormalFromObjectToWorldSpace(v2f(ns)));
        ns = (glm::dot(ns, ng) > 0.0f ? ns : -ns);
    } else
        ns = ng;

    Vec2 texCoord = { 0.0f, 0.0f };
    if(data->texCoord)
        texCoord = data->texCoord[idx.x] * u + data->texCoord[idx.y] * v +
            data->texCoord[idx.z] * w;

    Vec3 ori = f2v(optixGetWorldRayOrigin());
    Vec3 dir = f2v(optixGetWorldRayDirection());
    Vec3 hit = ori + optixGetRayTmin() * dir;
    builtinMaterialSample(data->material, payload, dir, hit, ng, ns, texCoord,
                          optixGetRayTime(), front);
}

GLOBAL void __anyhit__OAH() {}
