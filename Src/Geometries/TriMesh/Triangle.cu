#include "hip/hip_runtime.h"
#include "../../Shared/KernelShared.hpp"

/*
 * Copyright (c) 2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

//from cuda/intersection_refinement.h
// Plane intersection -- used for refining triangle hit points.  Note
// that this skips zero denom check (for rays perpindicular to plane normal)
// since we know that the ray intersects the plane.
static INLINEDEVICE float intersectPlane(const optix::float3 &origin,
    const optix::float3 &direction,
    const optix::float3 &normal,
    const optix::float3 &point) {
// Skipping checks for non-zero denominator since we know that ray intersects this plane
    return -(optix::dot(normal, origin - point)) / optix::dot(normal, direction);

}

// Offset the hit point using integer arithmetic
static INLINEDEVICE optix::float3 offset(const optix::float3 &hit_point, const optix::float3 &normal) {
    using namespace optix;

    const float epsilon = 1.0e-4f;
    const float offset = 4096.0f * 2.0f;

    float3 offset_point = hit_point;
    if ((__float_as_int(hit_point.x) & 0x7fffffff) < __float_as_int(epsilon)) {
        offset_point.x += epsilon * normal.x;
    }
    else {
        offset_point.x = __int_as_float(__float_as_int(offset_point.x) + int(copysign(offset, hit_point.x) * normal.x));
    }

    if ((__float_as_int(hit_point.y) & 0x7fffffff) < __float_as_int(epsilon)) {
        offset_point.y += epsilon * normal.y;
    }
    else {
        offset_point.y = __int_as_float(__float_as_int(offset_point.y) + int(copysign(offset, hit_point.y) * normal.y));
    }

    if ((__float_as_int(hit_point.z) & 0x7fffffff) < __float_as_int(epsilon)) {
        offset_point.z += epsilon * normal.z;
    }
    else {
        offset_point.z = __int_as_float(__float_as_int(offset_point.z) + int(copysign(offset, hit_point.z) * normal.z));
    }

    return offset_point;
}

// Refine the hit point to be more accurate and offset it for reflection and
// refraction ray starting points.
static INLINEDEVICE void refine_and_offset_hitpoint(const optix::float3 &original_hit_point, const optix::float3 &direction,
    const optix::float3 &normal, const optix::float3 &p, optix::float3 &back_hit_point,
    optix::float3 &front_hit_point) {
    using namespace optix;

    // Refine hit point
    float  refined_t = intersectPlane(original_hit_point, direction, normal, p);
    float3 refined_hit_point = original_hit_point + refined_t * direction;

    // Offset hit point
    if (dot(direction, normal) > 0.0f) {
        back_hit_point = offset(refined_hit_point, normal);
        front_hit_point = offset(refined_hit_point, -normal);
    }
    else {
        back_hit_point = offset(refined_hit_point, -normal);
        front_hit_point = offset(refined_hit_point, normal);
    }
}

rtDeclareVariable(Vec3, backHitPoint, attribute backHitPoint, );
rtDeclareVariable(Vec3, frontHitPoint, attribute frontHitPoint, );
rtDeclareVariable(Vec3, normal, attribute normal, );
rtDeclareVariable(Vec2, texCoord, attribute texCoord, );

rtBuffer<float3> geometryVertexBuffer;
rtBuffer<uint3> geometryIndexBuffer;
rtBuffer<Vec3> geometryNormalBuffer;
rtBuffer<Vec2> geometryTexCoordBuffer;

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

INLINEDEVICE void setAttributes(const uint3 &idx, const Vec3 &ng, 
    const Vec3 &hit, const Vec3 &refPoint, float u, float v, float w) {
    if (geometryNormalBuffer.size())
        normal = geometryNormalBuffer[idx.x] * u
        + geometryNormalBuffer[idx.y] * v
        + geometryNormalBuffer[idx.z] * w;
    else normal = normalize(ng);

    if (dot(ray.direction, normal) > 0.0f)
        normal = -normal;

    if (geometryTexCoordBuffer.size())
        texCoord = geometryTexCoordBuffer[idx.x] * u
        + geometryTexCoordBuffer[idx.y] * v
        + geometryTexCoordBuffer[idx.z] * w;
    else texCoord = make_float2(0.0f);

    refine_and_offset_hitpoint(hit, ray.direction, ng, refPoint,
        backHitPoint, frontHitPoint);
}

//from cuda/triangle_mesh.cu
RT_PROGRAM void meshAttributes() {
    const uint3 idx = geometryIndexBuffer[rtGetPrimitiveIndex()];
    const Vec3 v0 = geometryVertexBuffer[idx.x];
    const Vec3 v1 = geometryVertexBuffer[idx.y];
    const Vec3 v2 = geometryVertexBuffer[idx.z];
    const Vec3 ng = cross(v1 - v0, v2 - v0);

    const Vec2 bar = rtGetTriangleBarycentrics();
    float u = 1.0f - bar.x - bar.y, v = bar.x, w = bar.y;
    setAttributes(idx, ng, v0 * u + v1 * v + v2 * w, v0, u, v, w);
}

RT_PROGRAM void bounds(int primIdx, float result[6]) {
    const uint3 idx = geometryIndexBuffer[primIdx];
    optix::Aabb *aabb = reinterpret_cast<optix::Aabb *>(result);
    aabb->include(geometryVertexBuffer[idx.x]);
    aabb->include(geometryVertexBuffer[idx.y]);
    aabb->include(geometryVertexBuffer[idx.z]);
}

RT_PROGRAM void intersect(int primIdx) {
    const uint3 idx = geometryIndexBuffer[primIdx];

    const float3 p0 = geometryVertexBuffer[idx.x];
    const float3 p1 = geometryVertexBuffer[idx.y];
    const float3 p2 = geometryVertexBuffer[idx.z];

    // Intersect ray with triangle
    float3 n;
    float  t, beta, gamma;
    if (intersect_triangle(ray, p0, p1, p2, n, t, beta, gamma)) {
        if (rtPotentialIntersection(t)) {
            float alpha = 1.0f - beta - gamma;
            setAttributes(idx, n, ray.origin + t * ray.direction, p0, alpha, beta, gamma);
            rtReportIntersection(0);
        }
    }
}
