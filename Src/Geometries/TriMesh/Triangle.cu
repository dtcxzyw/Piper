#include "hip/hip_runtime.h"
#include "../../Shared/KernelShared.hpp"
#include "DataDesc.hpp"

GLOBAL void __closesthit__RCH() {
    const DataDesc* data = getSBTData<DataDesc>();
    float2 uv = optixGetTriangleBarycentrics();
    float u = uv.x, v = uv.y, w = 1.0f - u - v;
    Payload* payload = getPayload();
    Uint3 idx = data->index[optixGetPrimitiveIndex()];
    Vec3 p0 = data->vertex[0], p1 = data->vertex[1], p2 = data->vertex[2];
    Vec3 ng = glm::cross(p1 - p0, p2 - p0);
    bool front = optixIsTriangleFrontFaceHit();
    Vec3 ns;
    if(data->normal) {
        ns = data->normal[idx.x] * u + data->normal[idx.y] * v +
            data->normal[idx.z] * w;
        ns = (glm::dot(ns, ng) > 0.0f ? ns : -ns);
    } else
        ns = ng;

    Vec2 texCoord = { 0.0f, 0.0f };
    if(data->texCoord)
        texCoord = data->texCoord[idx.x] * u + data->texCoord[idx.y] * v +
            data->texCoord[idx.z] * w;

    Vec3 ori = f2v(optixGetWorldRayOrigin());
    Vec3 dir = f2v(optixGetWorldRayDirection());
    Vec3 hit = ori + optixGetRayTmin() * dir;
    builtinMaterialSample(data->material, payload, dir, hit, ng, ns, texCoord,
                          optixGetTime(), front);
    payload->hit = false;
    payload->rad = glm::normalize(glm::abs(ns));
}

GLOBAL void __anyhit__OAH() {}
