#include "hip/hip_runtime.h"
#include "../../Shared/KernelShared.hpp"
#include "DataDesc.hpp"

GLOBAL void __closesthit__RCH() {
    const DataDesc* data = getSBTData<DataDesc>();
    float2 uv = optixGetTriangleBarycentrics();
    float u = uv.x, v = uv.y, w = 1.0f - u - v;
    Payload* payload = getPayload();
    Uint3 idx = data->index[optixGetPrimitiveIndex()];
    Vec3 p0 = data->vertex[idx.x], p1 = data->vertex[idx.y],
         p2 = data->vertex[idx.z];
    Vec3 ng = glm::normalize(glm::cross(p1 - p0, p2 - p0));
    ng = f2v(optixTransformNormalFromObjectToWorldSpace(v2f(ng)));
    bool front = optixIsTriangleFrontFaceHit();
    Vec3 ns;
    if(data->normal) {
        ns = glm::normalize(data->normal[idx.x] * u + data->normal[idx.y] * v +
                            data->normal[idx.z] * w);
        ns = f2v(optixTransformNormalFromObjectToWorldSpace(v2f(ns)));
        ns = (glm::dot(ns, ng) > 0.0f ? ns : -ns);
    } else
        ns = ng;

    Vec2 texCoord = { 0.0f, 0.0f };
    if(data->texCoord)
        texCoord = data->texCoord[idx.x] * u + data->texCoord[idx.y] * v +
            data->texCoord[idx.z] * w;

    Vec3 ori = f2v(optixGetWorldRayOrigin());
    Vec3 dir = f2v(optixGetWorldRayDirection());
    Vec3 hit = ori + optixGetRayTmin() * dir;
    builtinMaterialSample(data->material, payload, dir, hit, ng, ns, texCoord,
                          optixGetRayTime(), front);
}

GLOBAL void __anyhit__OAH() {}
