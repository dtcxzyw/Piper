#include "hip/hip_runtime.h"
#include "../../Shared/KernelShared.hpp"
#include "DataDesc.hpp"

GLOBAL void __closesthit__RCH() {
    const DataDesc* data =
        reinterpret_cast<DataDesc*>(optixGetSbtDataPointer());
    float2 vw = optixGetTriangleBarycentrics();
    Payload* payload = getPayload();
    payload->hit = false;
    payload->rad = Spectrum{ 1.0f - vw.x - vw.y, vw.x, vw.y };
}

GLOBAL void __anyhit__OAH() {}
