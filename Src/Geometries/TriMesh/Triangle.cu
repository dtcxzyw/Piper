#include "hip/hip_runtime.h"
#include "DataDesc.hpp"
#include "../../CUDA.hpp"

rtDeclareVariable(optix::Ray, currentRay, rtCurrentRay, );
rtDeclareVariable(Vec3, shadingNormal, attribute shadingNormal, );
rtDeclareVariable(Vec3, geometricNormal, attribute geometricNormal, );
rtDeclareVariable(Vec2, texCoord, attribute texCoord, );
rtBuffer<Vertex> geometryVertex;
rtBuffer<uint3> geometryIndex;

RT_PROGRAM void intersect(int index) {
    uint3 idx = geometryIndex[index];
    Vertex v0 = geometryVertex[idx.x], v1 = geometryVertex[idx.y],
        v2 = geometryVertex[idx.z];
    Vec3 n;
    float t, beta, gamma;
    //TODO:early exit version
    bool res = optix::intersect_triangle_branchless(currentRay, v0.pos, v1.pos, v2.pos,
        n, t, beta, gamma);
    if (rtPotentialIntersection(t)) {
        geometricNormal = n;
        float alpha = 1.0f - (beta + gamma);
        shadingNormal = v0.normal * beta + v1.normal * gamma +
            v2.normal * alpha;
        texCoord = v0.texCoord * beta + v1.texCoord * gamma + v2.texCoord * alpha;
        rtReportIntersection(0);
    }
}

RT_PROGRAM void bounds(int index, float result[6]) {
    optix::Aabb &aabb = *reinterpret_cast<optix::Aabb *>(result);
    uint3 idx = geometryIndex[index];
    aabb.include(geometryVertex[idx.x].pos);
    aabb.include(geometryVertex[idx.y].pos);
    aabb.include(geometryVertex[idx.z].pos);
}
