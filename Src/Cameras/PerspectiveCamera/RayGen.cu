#include "hip/hip_runtime.h"
#include "../../Shared/KernelShared.hpp"

rtDeclareVariable(float3, cameraBase, , );
rtDeclareVariable(float3, cameraDown, , );
rtDeclareVariable(float3, cameraRight, , );
rtDeclareVariable(float3, cameraHole, , );
rtDeclareVariable(float3, cameraFStopX, , );
rtDeclareVariable(float3, cameraFStopY, , );
rtDeclareVariable(float, cameraFocal, , );
rtDeclareVariable(float3, cameraAxis, , );

DEVICE void sampleRay(uint2 pixel, RaySample &sample, uint32 &seed) {
    ++seed;
    Vec2 pixelPos = { sample1(seed) + pixel.x, sample2(seed) + pixel.y };
    Vec3 ori = cameraBase + cameraRight * pixelPos.x + cameraDown * pixelPos.y;
    Vec3 pinHoleDir = cameraHole - ori;
    Vec3 focalPoint = ori + pinHoleDir * (cameraFocal / dot(cameraAxis, pinHoleDir));
    float angle = twoPi * sample3(seed);
    float radius = sqrtf(sample4(seed));
    sample.ori = cameraHole + radius * cosf(angle) * cameraFStopX +
        radius * sinf(angle) * cameraFStopY;
    sample.dir = normalize(focalPoint - sample.ori);
}
