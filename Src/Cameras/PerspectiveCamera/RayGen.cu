#include "hip/hip_runtime.h"
#include "../../Shared/KernelShared.hpp"
#include "DataDesc.hpp"

DEVICE RaySample __direct_callable__sampleRay(float px, float py,
                                              SamplerContext& sampler) {
    auto data = getSBTData<DataDesc>();
    Vec3 ori = data->base + data->right * px + data->down * py;
    Vec3 pinHoleDir = data->hole - ori;
    Vec3 focalPoint =
        ori + pinHoleDir * (data->focal / dot(data->axis, pinHoleDir));
    float angle = glm::two_pi<float>() * sampler();
    float radius = sqrtf(sampler());
    RaySample res;
    res.ori = data->hole + radius * cosf(angle) * data->fStopX +
        radius * sinf(angle) * data->fStopY;
    res.dir = normalize(focalPoint - res.ori);
    return res;
}

void check(RayGenerateFunction = __direct_callable__sampleRay);
