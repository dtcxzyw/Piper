#include "hip/hip_runtime.h"
#include "../../Shared/KernelShared.hpp"
#include "DataDesc.hpp"

DEVICE RaySample __direct_callable__sampleRay(Uint2 pixel, uint32& seed) {
    ++seed;
    Vec2 pixelPos = { sample<0>(seed) + pixel.x, sample<1>(seed) + pixel.y };
    const KernelData* data =
        reinterpret_cast<KernelData*>(optixGetSbtDataPointer());
    Vec3 ori = data->base + data->right * pixelPos.x + data->down * pixelPos.y;
    Vec3 pinHoleDir = data->hole - ori;
    Vec3 focalPoint =
        ori + pinHoleDir * (data->focal / dot(data->axis, pinHoleDir));
    float angle = glm::two_pi<float>() * sample<2>(seed);
    float radius = sqrtf(sample<3>(seed));
    RaySample res;
    res.ori = data->hole + radius * cosf(angle) * data->fStopX +
        radius * sinf(angle) * data->fStopY;
    res.dir = normalize(focalPoint - res.ori);
    return res;
}
