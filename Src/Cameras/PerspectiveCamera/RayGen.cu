#include "hip/hip_runtime.h"
#include "../../CUDA.hpp"

rtDeclareVariable(float3, cameraBase, , );
rtDeclareVariable(float3, cameraDown, , );
rtDeclareVariable(float3, cameraRight, , );
rtDeclareVariable(float3, cameraHole, , );
rtDeclareVariable(float3, cameraFStopX, , );
rtDeclareVariable(float3, cameraFStopY, , );

DEVICE void sampleRay(uint2 pixel, RaySample &sample, uint32 &seed) {
    ++seed;
    float2 pixelPos = { sample1(seed)*pixel.x, sample2(seed)*pixel.y };
    sample.ori = cameraBase + cameraRight * pixelPos.x +
        cameraDown * pixelPos.y;
    float angle = twoPi * sample3(seed);
    float radius = sqrtf(sample4(seed));
    float3 holePos = cameraHole + radius * cosf(angle) * cameraFStopX +
        radius * sinf(angle) * cameraFStopY;
    sample.dir = normalize(holePos - sample.ori);
}
