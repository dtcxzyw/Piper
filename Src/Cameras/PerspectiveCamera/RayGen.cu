#include "hip/hip_runtime.h"
#include "../../Shared/KernelShared.hpp"
#include "DataDesc.hpp"

DEVICE RaySample __direct_callable__sampleRay(Vec2 pixelPos,
                                              SamplerContext& sampler) {
    auto data = getSBTData<DataDesc>();
    Vec3 ori = data->base + data->right * pixelPos.x + data->down * pixelPos.y;
    Vec3 pinHoleDir = data->hole - ori;
    Vec3 focalPoint =
        ori + pinHoleDir * (data->focal / dot(data->axis, pinHoleDir));
    float angle = glm::two_pi<float>() * sampler();
    float radius = sqrtf(sampler());
    RaySample res;
    res.ori = data->hole + radius * cosf(angle) * data->fStopX +
        radius * sinf(angle) * data->fStopY;
    res.dir = normalize(focalPoint - res.ori);
    return res;
}
