#include "hip/hip_runtime.h"
#include "../../Shared/KernelShared.hpp"
#include "DataDesc.hpp"

DEVICE Spectrum __continuation_callable__traceKernel(RaySample ray,
                                                     SamplerContext* sampler) {
    Spectrum res{ 0.0f };
    auto data = getSBTData<DataDesc>();
    Payload payload;
    payload.sampler = sampler;
    uint32_t p0, p1;
    packPointer(&payload, p0, p1);
    Spectrum att{ 1.0f };
    for(unsigned i = 0; i < data->maxDepth; ++i) {
        payload.hit = false;
        payload.f = Spectrum{ 0.0f };
        payload.rad = Spectrum{ 0.0f };
        // Russian roulette
        if(i > 3) {
            float q = fmax(0.05f,
                           1.0f -
                               (0.212671f * att.r + 0.715160f * att.g +
                                0.072169f * att.b));
            if((*sampler)() < q)
                break;
            att /= 1.0 - q;
        }
        // TODO:rayTime
        optixTrace(launchParam.root, v2f(ray.ori), v2f(ray.dir), eps, 1e20f,
                   0.0f, 255, OPTIX_RAY_FLAG_NONE, radianceOffset,
                   traceSBTStride, radianceMiss, p0, p1);
        res += att * payload.rad;
        if(!payload.hit)
            break;
        att *= payload.f;
        ray.ori = payload.ori;
        ray.dir = payload.wi;
    }
    return res;
}

void check(PixelSampleFunction = __continuation_callable__traceKernel) {}
