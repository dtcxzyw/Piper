#include "hip/hip_runtime.h"
#include "../../CUDA.hpp"

rtDeclareVariable(rtObject, globalTopNode, , );
rtDeclareVariable(unsigned, integratorMaxDepth, , );
rtDeclareVariable(unsigned, integratorSample, , );
rtBuffer<float4, 2> driverOutputBuffer;
rtDeclareVariable(uint2, driverBegin, , );
rtDeclareVariable(unsigned, driverIndex, , );
rtDeclareVariable(int, driverFiltBadColor, , );
rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );

DEVICE void sampleRay(uint2 pixel, RaySample &sample, uint32 &seed);

RT_PROGRAM void traceKernel() {
    Spectrum res = black;
    uint2 pixelPos = launchIndex;
    optix::size_t2 filmSize = driverOutputBuffer.size();
    uint32 id = filmSize.x * (driverIndex * filmSize.y + pixelPos.y) + pixelPos.x;
    for (uint32 i = 0; i < integratorSample; ++i) {
        uint32 seed = initSeed(id * integratorSample + i, 0xc3fea875);
        optix::Ray curRay;
        {
            RaySample sample;
            sampleRay(pixelPos, sample, seed);
            curRay = optix::make_Ray(sample.ori, sample.dir, 0, 0.0f, RT_DEFAULT_MAX);
        }
        Spectrum att = make_float3(1.0f);
        for (uint32 i = 0; i < integratorMaxDepth; ++i) {
            Payload payload;
            payload.index = seed;
            payload.hit = false;
            payload.f = black;
            payload.rad = black;
            rtTrace(globalTopNode, curRay, payload, geometryMask | lightVolumeMask);
            seed = payload.index;
            res += att * payload.rad;
            att *= payload.f;
            if (!payload.hit)
                break;
        }
    }
    if (driverFiltBadColor & !(isfinite(res.x) & isfinite(res.y) & isfinite(res.z)))
        return;
    driverOutputBuffer[pixelPos] +=
        make_float4(res, static_cast<float>(integratorSample));
}
