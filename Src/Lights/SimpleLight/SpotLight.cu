#include "hip/hip_runtime.h"
#include "../../Shared/KernelShared.hpp"
#include "DataDesc.hpp"

DEVICE LightSample __continuation_callable__sample(const Vec3& pos,
                                                   float rayTime,
                                                   SamplerContext& sampler) {
    auto light = getSBTData<SpotLightData>();
    Vec3 diff = light->pos - pos;
    float invSqrDis = 1.0f / dot(diff, diff);
    float invDis = sqrt(invSqrDis);
    float angle = dot(diff, light->negSpotDir) * invDis;
    if(angle <= light->outerCutOff) {
        LightSample res;
        res.rad = Spectrum{ 0.0f };
        return res;
    }
    unsigned noHit = 0;
    optixTrace(
        launchParam.root, v2f(pos), v2f(diff), eps, oneMinusEps, rayTime, 255,
        OPTIX_RAY_FLAG_DISABLE_ANYHIT | OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT |
            OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT,
        occlusionOffset, traceSBTStride, occlusionMiss, noHit);
    LightSample res;
    if(noHit) {
        res.rad = light->lum *
            (invSqrDis *
             (light->invDelta ?
                  fminf(1.0f, (angle - light->outerCutOff) * light->invDelta) :
                  1.0f));
    } else
        res.rad = Spectrum{ 0.0f };
    res.wi = diff * invDis;
    return res;
}

void check(LightSampleFunction = __continuation_callable__sample);
