#include "hip/hip_runtime.h"
#include "../../Shared/KernelShared.hpp"
#include "DataDesc.hpp"

DEVICE LightSample __continuation_callable__sample(const Vec3& pos,
                                                   float rayTime,
                                                   SamplerContext& sampler) {
    auto light = getSBTData<PointLightData>();
    Vec3 diff = light->pos - pos;
    unsigned noHit = 0;
    optixTrace(
        launchParam.root, v2f(pos), v2f(diff), eps, oneMinusEps, rayTime, 255,
        OPTIX_RAY_FLAG_DISABLE_ANYHIT | OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT |
            OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT,
        occlusionOffset, traceSBTStride, occlusionMiss, noHit);
    LightSample res;
    float invSqrDis = 1.0f / dot(diff, diff);
    res.rad = (noHit ? light->lum * invSqrDis : Spectrum{ 0.0f });
    res.wi = diff * sqrt(invSqrDis);
    return res;
}

void check(LightSampleFunction = __continuation_callable__sample);
