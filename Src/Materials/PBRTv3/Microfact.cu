#include "hip/hip_runtime.h"
#pragma once
#include "Microfact.hpp"
//Trowbridge-Reitz Distribution

namespace TrowbridgeReitzDistribution {
    DEVICE float roughnessToAlpha(float roughness) {
        roughness = fmaxf(roughness, 1e-3f);
        float x = logf(roughness);
        float res = 0.000640711f;
        res = res * x + 0.0171201f;
        res = res * x + 0.1734f;
        res = res * x + 0.819955f;
        return res * x + 1.62142f;
    }
    DEVICE float D(const Vec3 &wh, const Vec2 &alpha) {
        float tan2ThetaV = tan2Theta(wh);
        if (isinf(tan2ThetaV)) return 0.0f;
        float cos4Theta = sqr(cos2Theta(wh));
        float e = (cos2Phi(wh) / sqr(alpha.x) + sin2Phi(wh) / sqr(alpha.y)) * tan2ThetaV;
        return 1.0f / (pi * alpha.x * alpha.y * cos4Theta * sqr(1.0f + e));
    }
    DEVICE float lambda(const Vec3 &w, const Vec2 &alpha) {
        float absTanTheta = fabsf(tanTheta(w));
        if (isinf(absTanTheta)) return 0.0f;
        // Compute alpha for direction w
        float alphaW = sqrtf(cos2Phi(w) * sqr(alpha.x) + sin2Phi(w) * sqr(alpha.y));
        float alpha2Tan2Theta = sqr(alphaW * absTanTheta);
        return (-1.0f + sqrtf(1.0f + alpha2Tan2Theta)) * 0.5f;
    }
    DEVICE Vec2 trowbridgeReitzSample11(float cosTheta, float U1, float U2) {
        // special case (normal incidence)      
        if (cosTheta > 0.9999f) {
            float r = sqrtf(U1 / (1.0f - U1));
            float phi = twoPi * U2;
            return make_float2(cos(phi), sin(phi)) * r;
        }

        float sinTheta = cos2Sin(cosTheta);
        float tanTheta = sinTheta / cosTheta;
        float a = 1.0f / tanTheta;
        float G1 = 2.0f / (1.0f + sqrtf(1.0f + 1.0f / (a * a)));

        // sample slope_x
        float A = 2.0f * U1 / G1 - 1.0f, A2 = sqr(A);
        float tmp = fminf(1e10f, 1.0f / (A2 - 1.0f));
        float B = tanTheta, B2 = sqr(B);
        float D = sqrtf(fmaxf(B2 * tmp * tmp - (A2 - B2) * tmp, 0.0f));
        float Btmp = B * tmp;
        float slopeX1 = Btmp - D;
        float slopeX2 = Btmp + D;
        float slopeX = (A < 0.0f | slopeX2 > 1.0f / tanTheta) ? slopeX1 : slopeX2;

        // sample slope_y
        float S;
        if (U2 > 0.5f) {
            S = 1.0f;
            U2 = 2.0f * (U2 - 0.5f);
        }
        else {
            S = -1.0f;
            U2 = 2.0f * (0.5f - U2);
        }
        float z =
            (U2 * (U2 * (U2 * 0.27385f - 0.73369f) + 0.46341f)) /
            (U2 * (U2 * (U2 * 0.093073f + 0.309420f) - 1.000000f) + 0.597999f);
        float slopeY = S * z * sqrtf(1.0f + sqr(slopeX));
        return make_float2(slopeX, slopeY);
    }

    DEVICE Vec3 trowbridgeReitzSample(const Vec3 &wi, const Vec2 &alpha,
        const Vec2 &u) {
// 1. stretch wi
        Vec3 wiStretched =
            normalize(make_float3(alpha.x * wi.x, alpha.y * wi.y, wi.z));

        // 2. simulate P22_{wi}(x_slope, y_slope, 1, 1)

        Vec2 slope = trowbridgeReitzSample11(cosTheta(wiStretched), u.x, u.y);

        // 3. rotate    
        float sinW = sinPhi(wiStretched), cosW = cosPhi(wiStretched);
        float tmpX = cosW * slope.x - sinW * slope.y;
        slope.y = sinW * slope.x + cosW * slope.y;
        slope.x = tmpX;

        // 4. unstretch
        slope *= alpha;

        // 5. compute normal
        return normalize(make_float3(-slope.x, -slope.y, 1.0f));
    }

    DEVICE Vec3 sampleWh(const Vec3 &wo, const Vec2 &alpha, const Vec2 &u) {
        Vec3 wh = trowbridgeReitzSample(wo.z >= 0.0f ? wo : -wo, alpha, u);
        return wo.z >= 0.0f ? wh : -wh;
    }

    DEVICE float pdf(const Vec3 &wo, const Vec3 &wh, const Vec2 &alpha) {
        float G1 = 1.0f / (1.0f + lambda(wo, alpha));
        return D(wh, alpha) * G1 * fabsf(dot(wo, wh)) / absCosTheta(wo);
    }
    DEVICE float G(const Vec3 &wo, const Vec3 &wi, const Vec2 &alpha) {
        return 1.0f / (1.0f + lambda(wo, alpha) + lambda(wi, alpha));
    }
}
