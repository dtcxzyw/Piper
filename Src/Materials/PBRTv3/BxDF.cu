#include "hip/hip_runtime.h"
#include "BxDF.hpp"
#include "Microfact.hpp"

DEVICE Spectrum LambertianReflection::f(const Vec3&, const Vec3&) const {
    return mR * invPi;
}

DEVICE Vec3 LambertianReflection::sampleF(const Vec3& wo, const Vec2& u) const {
    Vec3 wi;
    optix::cosine_sample_hemisphere(u.x, u.y, wi);
    wi.z = copysignf(wi.z, wo.z);
    return wi;
}

DEVICE float LambertianReflection::pdf(const Vec3& wo, const Vec3& wi) const {
    return sameHemisphere(wo, wi) ? absCosTheta(wi) * invPi : 0.0f;
}

DEVICE Spectrum MicrofacetReflection::f(const Vec3& wo, const Vec3& wi) const {
    float cosThetaO = absCosTheta(wo), cosThetaI = absCosTheta(wi);
    Vec3 wh = wi + wo;
    // Handle degenerate cases for microfacet reflection
    if(fminf(cosThetaI, cosThetaO) < eps)
        return black;
    if(fmaxf(wh) < eps)
        return black;
    wh = normalize(wh);
    Spectrum F = mFresnel.eval(dot(wi, wh));
    return mR * F *
        (TrowbridgeReitzDistribution::D(wh, mAlpha) *
         TrowbridgeReitzDistribution::G(wo, wi, mAlpha) /
         (4.0f * cosThetaI * cosThetaO));
}

DEVICE Vec3 MicrofacetReflection::sampleF(const Vec3& wo, const Vec2& u) const {
    // Sample microfacet orientation $\wh$ and reflected direction $\wi$
    // if (wo.z <= eps) return {};
    Vec3 wh = TrowbridgeReitzDistribution::sampleWh(wo, mAlpha, u);
    Vec3 wi = reflect(wo, wh);
    return wi;
}

DEVICE float MicrofacetReflection::pdf(const Vec3& wo, const Vec3& wi) const {
    if(!sameHemisphere(wo, wi))
        return 0.0f;
    Vec3 wh = normalize(wo + wi);
    return TrowbridgeReitzDistribution::pdf(wo, wh, mAlpha) /
        (4.0f * dot(wo, wh));
}

DEVICE Vec2 MicrofacetReflection::toAlpha(const Vec2& roughness) {
    using namespace TrowbridgeReitzDistribution;
    return make_float2(roughnessToAlpha(roughness.x),
                       roughnessToAlpha(roughness.y));
}
