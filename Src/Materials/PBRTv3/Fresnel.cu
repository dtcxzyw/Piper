#include "hip/hip_runtime.h"
#include "Fresnel.hpp"
DEVICE Spectrum FresnelDielectric::eval(float cosThetaI) const {
    float etaI = mEtaI, etaT = mEtaT;
    if(cosThetaI < 0.0f) {
        cosThetaI = -cosThetaI;
        swap(etaI, etaT);
    }
    float sinThetaT = etaI / etaT * cos2Sin(cosThetaI);

    // Total internal reflection
    if(sinThetaT >= 1.0f)
        return make_float3(1.0f);

    float cosThetaT = sin2Cos(sinThetaT);

    float ii = etaI * cosThetaI, it = etaI * cosThetaT;
    float ti = etaT * cosThetaI, tt = etaT * cosThetaT;

    float a = (ti - it) / (ti + it);
    float b = (ii - tt) / (ii + tt);

    return make_float3(0.5f * (a * a + b * b));
}

// https://seblagarde.wordpress.com/2013/04/29/memo-on-fresnel-equations/
DEVICE Spectrum FresnelConductor::eval(float cosThetaI) const {
    cosThetaI = fabsf(cosThetaI);
    Spectrum eta = mEtaT / mEtaI;
    Spectrum etak = mK / mEtaI;

    float cosThetaI2 = cosThetaI * cosThetaI;
    float sinThetaI2 = 1.0f - cosThetaI2;
    Spectrum eta2 = eta * eta;
    Spectrum etak2 = etak * etak;

    Spectrum t0 = eta2 - etak2 - sinThetaI2;
    Spectrum a2plusb2 = sqrtf(t0 * t0 + 4.0f * eta2 * etak2);
    Spectrum t1 = a2plusb2 + cosThetaI2;
    Spectrum a = sqrtf(0.5f * (a2plusb2 + t0));
    Spectrum t2 = 2.0f * cosThetaI * a;
    Spectrum Rs = (t1 - t2) / (t1 + t2);

    Spectrum t3 = cosThetaI2 * a2plusb2 + sinThetaI2 * sinThetaI2;
    Spectrum t4 = t2 * sinThetaI2;
    Spectrum Rp = Rs * (t3 - t4) / (t3 + t4);

    return 0.5f * (Rp + Rs);
}
