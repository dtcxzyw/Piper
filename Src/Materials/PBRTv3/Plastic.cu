#include "hip/hip_runtime.h"
#include "../../CUDA.hpp"
#include "BxDF.hpp"

rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(Vec2, texCoord, attribute texCoord, );
TextureSampler(float4) materialKd;
TextureSampler(float4) materialKs;
TextureSampler(float2) materialRoughness;

RT_PROGRAM void closestHit() {
    ShadingSpace ss = calcPayload();
    Spectrum Kd = make_float3(tex2D(materialKd, texCoord.x, texCoord.y));
    Spectrum Ks = make_float3(tex2D(materialKs, texCoord.x, texCoord.y));
    Vec2 roughness = tex2D(materialRoughness, texCoord.x, texCoord.y);
    LambertianReflection lr(Kd);
    MicrofacetReflection mr(Ks, roughness, FresnelDielectric(1.5f, 1.0f));
    uint32 seed = ++payload.index;
    bool choice = sample1(seed) < 0.5f;
    Vec2 u = make_float2(sample2(seed), sample3(seed));
    Vec3 wi = choice ? lr.sampleF(ss.wo, u) : mr.sampleF(ss.wo, u);
    payload.f = (lr.f(ss.wo, wi) + mr.f(ss.wo, wi)) *
        (fabsf(dot(wi, ss.base.m_normal)) /
        (0.5f * (lr.pdf(ss.wo, wi) + mr.pdf(ss.wo, wi))));
    payload.wi = wi;
    ss.base.inverse_transform(payload.wi);
    LightSample ls = sampleOneLight();
    wi = ss.toLocal(ls.wi);
    payload.rad = ls.rad * (lr.f(ss.wo, wi) + mr.f(ss.wo, wi)) *
        (fabsf(dot(wi, ss.base.m_normal)) /
        (0.5f * (lr.pdf(ss.wo, wi) + mr.pdf(ss.wo, wi))));
}
