#include "hip/hip_runtime.h"
#include "../../Shared/KernelShared.hpp"
#include "DataDesc.hpp"

DEVICE void bsdf_init(MDL::Shading_state_material* state,
                      const MDL::Resource_data* res_data,
                      const void* exception_state, const char* arg_block_data);
DEVICE void bsdf_sample(MDL::Bsdf_sample_data* data,
                        MDL::Shading_state_material* state,
                        const MDL::Resource_data* res_data,
                        const void* exception_state,
                        const char* arg_block_data);
DEVICE void bsdf_evaluate(MDL::Bsdf_evaluate_data* data,
                          MDL::Shading_state_material* state,
                          const MDL::Resource_data* res_data,
                          const void* exception_state,
                          const char* arg_block_data);
DEVICE void bsdf_pdf(MDL::Bsdf_pdf_data* data,
                     MDL::Shading_state_material* state,
                     const MDL::Resource_data* res_data,
                     const void* exception_state, const char* arg_block_data);

// From examples/mdl_sdk/shared/texture_support_cuda.h

// Stores a float4 in a float[4] array.
INLINEDEVICE void store_result4(float res[4], const float4& v) {
    res[0] = v.x;
    res[1] = v.y;
    res[2] = v.z;
    res[3] = v.w;
}

// Stores a float in all elements of a float[4] array.
INLINEDEVICE void store_result4(float res[4], float s) {
    res[0] = res[1] = res[2] = res[3] = s;
}

// Stores the given float values in a float[4] array.
INLINEDEVICE void store_result4(float res[4], float v0, float v1, float v2,
                                float v3) {
    res[0] = v0;
    res[1] = v1;
    res[2] = v2;
    res[3] = v3;
}

// Stores a float3 in a float[3] array.
INLINEDEVICE void store_result3(float res[3], float3 const& v) {
    res[0] = v.x;
    res[1] = v.y;
    res[2] = v.z;
}

// Stores a float4 in a float[3] array, ignoring v.w.
INLINEDEVICE void store_result3(float res[3], const float4& v) {
    res[0] = v.x;
    res[1] = v.y;
    res[2] = v.z;
}

// Stores a float in all elements of a float[3] array.
INLINEDEVICE void store_result3(float res[3], float s) {
    res[0] = res[1] = res[2] = s;
}

// Stores the given float values in a float[3] array.
INLINEDEVICE void store_result3(float res[3], float v0, float v1, float v2) {
    res[0] = v0;
    res[1] = v1;
    res[2] = v2;
}

// Stores the luminance if a given float[3] in a float.
INLINEDEVICE void store_result1(float* res, float3 const& v) {
    // store luminance
    *res = 0.212671 * v.x + 0.715160 * v.y + 0.072169 * v.z;
}

// Stores the luminance if a given float[3] in a float.
INLINEDEVICE void store_result1(float* res, float v0, float v1, float v2) {
    // store luminance
    *res = 0.212671 * v0 + 0.715160 * v1 + 0.072169 * v2;
}

// Stores a given float in a float
INLINEDEVICE void store_result1(float* res, float s) {
    *res = s;
}

// ------------------------------------------------------------------------------------------------
// Textures
// ------------------------------------------------------------------------------------------------

// Applies wrapping and cropping to the given coordinate.
// Note: This macro returns if wrap mode is clip and the coordinate is out of
// range.
#define WRAP_AND_CROP_OR_RETURN_BLACK(val, inv_dim, wrap_mode, crop_vals,   \
                                      store_res_func)                       \
    do {                                                                    \
        if((wrap_mode) == mi::neuraylib::TEX_WRAP_REPEAT &&                 \
           (crop_vals)[0] == 0.0f && (crop_vals)[1] == 1.0f) {              \
            /* Do nothing, use texture sampler default behavior */          \
        } else {                                                            \
            if((wrap_mode) == mi::neuraylib::TEX_WRAP_REPEAT)               \
                val = val - floorf(val);                                    \
            else {                                                          \
                if((wrap_mode) == mi::neuraylib::TEX_WRAP_CLIP &&           \
                   (val < 0.0f || val >= 1.0f)) {                           \
                    store_res_func(result, 0.0f);                           \
                    return;                                                 \
                } else if((wrap_mode) ==                                    \
                          mi::neuraylib::TEX_WRAP_MIRRORED_REPEAT) {        \
                    float floored_val = floorf(val);                        \
                    if((int(floored_val) & 1) != 0)                         \
                        val = 1.0f - (val - floored_val);                   \
                    else                                                    \
                        val = val - floored_val;                            \
                }                                                           \
                float inv_hdim = 0.5f * (inv_dim);                          \
                val = fminf(fmaxf(val, inv_hdim), 1.f - inv_hdim);          \
            }                                                               \
            val = val * ((crop_vals)[1] - (crop_vals)[0]) + (crop_vals)[0]; \
        }                                                                   \
    } while(0)

#define USE_SMOOTHERSTEP_FILTER
#ifdef USE_SMOOTHERSTEP_FILTER
// Modify texture coordinates to get better texture filtering,
// see http://www.iquilezles.org/www/articles/texture/texture.htm
#define APPLY_SMOOTHERSTEP_FILTER()                                \
    do {                                                           \
        u = u * tex.size.x + 0.5f;                                 \
        v = v * tex.size.y + 0.5f;                                 \
                                                                   \
        float u_i = floorf(u), v_i = floorf(v);                    \
        float u_f = u - u_i;                                       \
        float v_f = v - v_i;                                       \
        u_f = u_f * u_f * u_f * (u_f * (u_f * 6.f - 15.f) + 10.f); \
        v_f = v_f * v_f * v_f * (v_f * (v_f * 6.f - 15.f) + 10.f); \
        u = u_i + u_f;                                             \
        v = v_i + v_f;                                             \
                                                                   \
        u = (u - 0.5f) * tex.inv_size.x;                           \
        v = (v - 0.5f) * tex.inv_size.y;                           \
    } while(0)
#else
#define APPLY_SMOOTHERSTEP_FILTER()
#endif

// Implementation of tex::lookup_float4() for a texture_2d texture.
DEVICE void tex_lookup_float4_2d(float result[4],
                                 Texture_handler_base const* self_base,
                                 unsigned texture_idx, float const coord[2],
                                 Tex_wrap_mode const wrap_u,
                                 Tex_wrap_mode const wrap_v,
                                 float const crop_u[2], float const crop_v[2]) {
    Texture_handler const* self =
        static_cast<Texture_handler const*>(self_base);

    if(texture_idx == 0 || texture_idx - 1 >= self->num_textures) {
        // invalid texture returns zero
        store_result4(result, 0.0f);
        return;
    }

    Texture const& tex = self->textures[texture_idx - 1];
    float u = coord[0], v = coord[1];
    WRAP_AND_CROP_OR_RETURN_BLACK(u, tex.inv_size.x, wrap_u, crop_u,
                                  store_result4);
    WRAP_AND_CROP_OR_RETURN_BLACK(v, tex.inv_size.y, wrap_v, crop_v,
                                  store_result4);

    APPLY_SMOOTHERSTEP_FILTER();

    store_result4(result, tex2D<float4>(tex.filtered_object, u, v));
}

// Implementation of tex::lookup_float3() for a texture_2d texture.
DEVICE void tex_lookup_float3_2d(float result[3],
                                 Texture_handler_base const* self_base,
                                 unsigned texture_idx, float const coord[2],
                                 Tex_wrap_mode const wrap_u,
                                 Tex_wrap_mode const wrap_v,
                                 float const crop_u[2], float const crop_v[2]) {
    Texture_handler const* self =
        static_cast<Texture_handler const*>(self_base);

    if(texture_idx == 0 || texture_idx - 1 >= self->num_textures) {
        // invalid texture returns zero
        store_result3(result, 0.0f);
        return;
    }

    Texture const& tex = self->textures[texture_idx - 1];
    float u = coord[0], v = coord[1];
    WRAP_AND_CROP_OR_RETURN_BLACK(u, tex.inv_size.x, wrap_u, crop_u,
                                  store_result3);
    WRAP_AND_CROP_OR_RETURN_BLACK(v, tex.inv_size.y, wrap_v, crop_v,
                                  store_result3);

    APPLY_SMOOTHERSTEP_FILTER();

    store_result3(result, tex2D<float4>(tex.filtered_object, u, v));
}

// Implementation of tex::texel_float4() for a texture_2d texture.
// Note: uvtile textures are not supported
DEVICE void tex_texel_float4_2d(float result[4],
                                Texture_handler_base const* self_base,
                                unsigned texture_idx, int const coord[2],
                                int const /*uv_tile*/[2]) {
    Texture_handler const* self =
        static_cast<Texture_handler const*>(self_base);

    if(texture_idx == 0 || texture_idx - 1 >= self->num_textures) {
        // invalid texture returns zero
        store_result4(result, 0.0f);
        return;
    }

    Texture const& tex = self->textures[texture_idx - 1];

    store_result4(result,
                  tex2D<float4>(tex.unfiltered_object,
                                float(coord[0]) * tex.inv_size.x,
                                float(coord[1]) * tex.inv_size.y));
}

// Implementation of tex::lookup_float4() for a texture_3d texture.
DEVICE void tex_lookup_float4_3d(float result[4],
                                 Texture_handler_base const* self_base,
                                 unsigned texture_idx, float const coord[3],
                                 Tex_wrap_mode wrap_u, Tex_wrap_mode wrap_v,
                                 Tex_wrap_mode wrap_w, float const crop_u[2],
                                 float const crop_v[2], float const crop_w[2]) {
    Texture_handler const* self =
        static_cast<Texture_handler const*>(self_base);

    if(texture_idx == 0 || texture_idx - 1 >= self->num_textures) {
        // invalid texture returns zero
        store_result4(result, 0.0f);
        return;
    }

    Texture const& tex = self->textures[texture_idx - 1];

    float u = coord[0], v = coord[1], w = coord[2];
    WRAP_AND_CROP_OR_RETURN_BLACK(u, tex.inv_size.x, wrap_u, crop_u,
                                  store_result4);
    WRAP_AND_CROP_OR_RETURN_BLACK(v, tex.inv_size.y, wrap_v, crop_v,
                                  store_result4);
    WRAP_AND_CROP_OR_RETURN_BLACK(w, tex.inv_size.z, wrap_w, crop_w,
                                  store_result4);

    store_result4(result, tex3D<float4>(tex.filtered_object, u, v, w));
}

// Implementation of tex::lookup_float3() for a texture_3d texture.
DEVICE void tex_lookup_float3_3d(float result[3],
                                 Texture_handler_base const* self_base,
                                 unsigned texture_idx, float const coord[3],
                                 Tex_wrap_mode wrap_u, Tex_wrap_mode wrap_v,
                                 Tex_wrap_mode wrap_w, float const crop_u[2],
                                 float const crop_v[2], float const crop_w[2]) {
    Texture_handler const* self =
        static_cast<Texture_handler const*>(self_base);

    if(texture_idx == 0 || texture_idx - 1 >= self->num_textures) {
        // invalid texture returns zero
        store_result3(result, 0.0f);
        return;
    }

    Texture const& tex = self->textures[texture_idx - 1];

    float u = coord[0], v = coord[1], w = coord[2];
    WRAP_AND_CROP_OR_RETURN_BLACK(u, tex.inv_size.x, wrap_u, crop_u,
                                  store_result3);
    WRAP_AND_CROP_OR_RETURN_BLACK(v, tex.inv_size.y, wrap_v, crop_v,
                                  store_result3);
    WRAP_AND_CROP_OR_RETURN_BLACK(w, tex.inv_size.z, wrap_w, crop_w,
                                  store_result3);

    store_result3(result, tex3D<float4>(tex.filtered_object, u, v, w));
}

// Implementation of tex::texel_float4() for a texture_3d texture.
DEVICE void tex_texel_float4_3d(float result[4],
                                Texture_handler_base const* self_base,
                                unsigned texture_idx, const int coord[3]) {
    Texture_handler const* self =
        static_cast<Texture_handler const*>(self_base);

    if(texture_idx == 0 || texture_idx - 1 >= self->num_textures) {
        // invalid texture returns zero
        store_result4(result, 0.0f);
        return;
    }

    Texture const& tex = self->textures[texture_idx - 1];

    store_result4(result,
                  tex3D<float4>(tex.unfiltered_object,
                                float(coord[0]) * tex.inv_size.x,
                                float(coord[1]) * tex.inv_size.y,
                                float(coord[2]) * tex.inv_size.z));
}

// Implementation of tex::lookup_float4() for a texture_cube texture.
DEVICE void tex_lookup_float4_cube(float result[4],
                                   Texture_handler_base const* self_base,
                                   unsigned texture_idx, float const coord[3]) {
    Texture_handler const* self =
        static_cast<Texture_handler const*>(self_base);

    if(texture_idx == 0 || texture_idx - 1 >= self->num_textures) {
        // invalid texture returns zero
        store_result4(result, 0.0f);
        return;
    }

    Texture const& tex = self->textures[texture_idx - 1];

    store_result4(
        result,
        texCubemap<float4>(tex.filtered_object, coord[0], coord[1], coord[2]));
}

// Implementation of tex::lookup_float3() for a texture_cube texture.
DEVICE void tex_lookup_float3_cube(float result[3],
                                   Texture_handler_base const* self_base,
                                   unsigned texture_idx, float const coord[3]) {
    Texture_handler const* self =
        static_cast<Texture_handler const*>(self_base);

    if(texture_idx == 0 || texture_idx - 1 >= self->num_textures) {
        // invalid texture returns zero
        store_result3(result, 0.0f);
        return;
    }

    Texture const& tex = self->textures[texture_idx - 1];

    store_result3(
        result,
        texCubemap<float4>(tex.filtered_object, coord[0], coord[1], coord[2]));
}

// Implementation of resolution_2d function needed by generated code.
// Note: uvtile textures are not supported
DEVICE void tex_resolution_2d(int result[2],
                              Texture_handler_base const* self_base,
                              unsigned texture_idx, int const /*uv_tile*/[2]) {
    Texture_handler const* self =
        static_cast<Texture_handler const*>(self_base);

    if(texture_idx == 0 || texture_idx - 1 >= self->num_textures) {
        // invalid texture returns zero
        result[0] = 0;
        result[1] = 0;
        return;
    }

    Texture const& tex = self->textures[texture_idx - 1];
    result[0] = tex.size.x;
    result[1] = tex.size.y;
}

// Implementation of resolution_3d function needed by generated code.
// Note: 3d textures are not supported
DEVICE void tex_resolution_3d(int result[3],
                              Texture_handler_base const* self_base,
                              unsigned texture_idx) {
    // invalid texture returns zero
    result[0] = 0;
    result[1] = 0;
    result[2] = 0;
}

// Implementation of texture_isvalid().
DEVICE bool tex_texture_isvalid(Texture_handler_base const* self_base,
                                unsigned texture_idx) {
    Texture_handler const* self =
        static_cast<Texture_handler const*>(self_base);

    return texture_idx != 0 && texture_idx - 1 < self->num_textures;
}

// ------------------------------------------------------------------------------------------------
// Light Profiles
// ------------------------------------------------------------------------------------------------

// Implementation of light_profile_power() for a light profile.
DEVICE float df_light_profile_power(Texture_handler_base const* self_base,
                                    unsigned light_profile_idx) {
    Texture_handler const* self =
        static_cast<Texture_handler const*>(self_base);
    if(light_profile_idx == 0 ||
       light_profile_idx - 1 >= self->num_lightprofiles)
        return 0.0f;  // invalid light profile returns zero

    const Lightprofile& lp = self->lightprofiles[light_profile_idx - 1];
    return lp.total_power;
}

// Implementation of light_profile_maximum() for a light profile.
DEVICE float df_light_profile_maximum(Texture_handler_base const* self_base,
                                      unsigned light_profile_idx) {
    Texture_handler const* self =
        static_cast<Texture_handler const*>(self_base);
    if(light_profile_idx == 0 ||
       light_profile_idx - 1 >= self->num_lightprofiles)
        return 0.0f;  // invalid light profile returns zero

    const Lightprofile& lp = self->lightprofiles[light_profile_idx - 1];
    return lp.candela_multiplier;
}

// Implementation of light_profile_isvalid() for a light profile.
DEVICE bool df_light_profile_isvalid(Texture_handler_base const* self_base,
                                     unsigned light_profile_idx) {
    Texture_handler const* self =
        static_cast<Texture_handler const*>(self_base);
    return light_profile_idx != 0 &&
        light_profile_idx - 1 < self->num_lightprofiles;
}

// binary search through CDF
INLINEDEVICE unsigned sample_cdf(const float* cdf, unsigned cdf_size,
                                 float xi) {
    unsigned li = 0;
    unsigned ri = cdf_size - 1;
    unsigned m = (li + ri) / 2;
    while(ri > li) {
        if(xi < cdf[m])
            ri = m;
        else
            li = m + 1;

        m = (li + ri) / 2;
    }

    return m;
}

// Implementation of df::light_profile_evaluate() for a light profile.
DEVICE float df_light_profile_evaluate(Texture_handler_base const* self_base,
                                       unsigned light_profile_idx,
                                       float const theta_phi[2]) {
    Texture_handler const* self =
        static_cast<Texture_handler const*>(self_base);
    if(light_profile_idx == 0 ||
       light_profile_idx - 1 >= self->num_lightprofiles)
        return 0.0f;  // invalid light profile returns zero

    const Lightprofile& lp = self->lightprofiles[light_profile_idx - 1];

    // map theta to 0..1 range
    const float u = (theta_phi[0] - lp.theta_phi_start.x) *
        lp.theta_phi_inv_delta.x / float(lp.angular_resolution.x - 1);

    // converting input phi from -pi..pi to 0..2pi
    float phi = (theta_phi[1] > 0.0f) ? theta_phi[1] :
                                        (float(2.0 * M_PI) + theta_phi[1]);

    // floorf wraps phi range into 0..2pi
    phi = phi - lp.theta_phi_start.y -
        floorf((phi - lp.theta_phi_start.y) * float(0.5 / M_PI)) *
            float(2.0 * M_PI);

    // (phi < 0.0f) is no problem, this is handle by the (black) border
    // since it implies lp.theta_phi_start.y > 0 (and we really have "no data"
    // below that)
    const float v =
        phi * lp.theta_phi_inv_delta.y / float(lp.angular_resolution.y - 1);

    // wrap_mode: border black would be an alternative (but it produces
    // artifacts at low res)
    if(u < 0.0f || u > 1.0f || v < 0.0f || v > 1.0f)
        return 0.0f;

    return tex2D<float>(lp.eval_data, u, v) * lp.candela_multiplier;
}

// Implementation of df::light_profile_sample() for a light profile.
DEVICE void df_light_profile_sample(float result[3],  // output: theta, phi, pdf
                                    Texture_handler_base const* self_base,
                                    unsigned light_profile_idx,
                                    float const xi[3])  // uniform random values
{
    result[0] = -1.0f;  // negative theta means no emission
    result[1] = -1.0f;
    result[2] = 0.0f;

    Texture_handler const* self =
        static_cast<Texture_handler const*>(self_base);
    if(light_profile_idx == 0 ||
       light_profile_idx - 1 >= self->num_lightprofiles)
        return;  // invalid light profile returns zero

    const Lightprofile& lp = self->lightprofiles[light_profile_idx - 1];
    uint2 res = lp.angular_resolution;

    // sample theta_out
    //-------------------------------------------
    float xi0 = xi[0];
    const float* cdf_data_theta = lp.cdf_data;  // CDF theta
    unsigned idx_theta =
        sample_cdf(cdf_data_theta, res.x - 1, xi0);  // binary search

    float prob_theta = cdf_data_theta[idx_theta];
    if(idx_theta > 0) {
        const float tmp = cdf_data_theta[idx_theta - 1];
        prob_theta -= tmp;
        xi0 -= tmp;
    }
    xi0 /= prob_theta;  // rescale for re-usage

    // sample phi_out
    //-------------------------------------------
    float xi1 = xi[1];
    const float* cdf_data_phi = cdf_data_theta + (res.x - 1)  // CDF theta block
        + (idx_theta * (res.y - 1));  // selected CDF for phi

    const unsigned idx_phi =
        sample_cdf(cdf_data_phi, res.y - 1, xi1);  // binary search
    float prob_phi = cdf_data_phi[idx_phi];
    if(idx_phi > 0) {
        const float tmp = cdf_data_phi[idx_phi - 1];
        prob_phi -= tmp;
        xi1 -= tmp;
    }
    xi1 /= prob_phi;  // rescale for re-usage

    // compute theta and phi
    //-------------------------------------------
    // sample uniformly within the patch (grid cell)
    const float2 start = lp.theta_phi_start;
    const float2 delta = lp.theta_phi_delta;

    const float cos_theta_0 = cosf(start.x + float(idx_theta) * delta.x);
    const float cos_theta_1 = cosf(start.x + float(idx_theta + 1u) * delta.x);

    //               n = \int_{\theta_0}^{\theta_1} \sin{\theta} \delta \theta
    //                 = 1 / (\cos{\theta_0} - \cos{\theta_1})
    //
    //             \xi = n * \int_{\theta_0}^{\theta_1} \sin{\theta} \delta
    //             \theta
    // => \cos{\theta} = (1 - \xi) \cos{\theta_0} + \xi \cos{\theta_1}

    const float cos_theta = (1.0f - xi1) * cos_theta_0 + xi1 * cos_theta_1;
    result[0] = acosf(cos_theta);
    result[1] = start.y + (float(idx_phi) + xi0) * delta.y;

    // align phi
    if(result[1] > float(2.0 * M_PI))
        result[1] -= float(2.0 * M_PI);  // wrap
    if(result[1] > float(1.0 * M_PI))
        result[1] = float(-2.0 * M_PI) + result[1];  // to [-pi, pi]

    // compute pdf
    //-------------------------------------------
    result[2] = prob_theta * prob_phi / (delta.y * (cos_theta_0 - cos_theta_1));
}

// Implementation of df::light_profile_pdf() for a light profile.
DEVICE float df_light_profile_pdf(Texture_handler_base const* self_base,
                                  unsigned light_profile_idx,
                                  float const theta_phi[2]) {
    Texture_handler const* self =
        static_cast<Texture_handler const*>(self_base);
    if(light_profile_idx == 0 ||
       light_profile_idx - 1 >= self->num_lightprofiles)
        return 0.0f;  // invalid light profile returns zero

    const Lightprofile& lp = self->lightprofiles[light_profile_idx - 1];

    // CDF data
    const uint2 res = lp.angular_resolution;
    const float* cdf_data_theta = lp.cdf_data;

    // map theta to 0..1 range
    const float theta = theta_phi[0] - lp.theta_phi_start.x;
    const int idx_theta = int(theta * lp.theta_phi_inv_delta.x);

    // converting input phi from -pi..pi to 0..2pi
    float phi = (theta_phi[1] > 0.0f) ? theta_phi[1] :
                                        (float(2.0 * M_PI) + theta_phi[1]);

    // floorf wraps phi range into 0..2pi
    phi = phi - lp.theta_phi_start.y -
        floorf((phi - lp.theta_phi_start.y) * float(0.5 / M_PI)) *
            float(2.0 * M_PI);

    // (phi < 0.0f) is no problem, this is handle by the (black) border
    // since it implies lp.theta_phi_start.y > 0 (and we really have "no data"
    // below that)
    const int idx_phi = int(phi * lp.theta_phi_inv_delta.y);

    // wrap_mode: border black would be an alternative (but it produces
    // artifacts at low res)
    if(idx_theta < 0 || idx_theta > (res.x - 2) || idx_phi < 0 ||
       idx_phi > (res.x - 2))
        return 0.0f;

    // get probability for theta
    //-------------------------------------------

    float prob_theta = cdf_data_theta[idx_theta];
    if(idx_theta > 0) {
        const float tmp = cdf_data_theta[idx_theta - 1];
        prob_theta -= tmp;
    }

    // get probability for phi
    //-------------------------------------------
    const float* cdf_data_phi = cdf_data_theta + (res.x - 1)  // CDF theta block
        + (idx_theta * (res.y - 1));  // selected CDF for phi

    float prob_phi = cdf_data_phi[idx_phi];
    if(idx_phi > 0) {
        const float tmp = cdf_data_phi[idx_phi - 1];
        prob_phi -= tmp;
    }

    // compute probability to select a position in the sphere patch
    const float2 start = lp.theta_phi_start;
    const float2 delta = lp.theta_phi_delta;

    const float cos_theta_0 = cos(start.x + float(idx_theta) * delta.x);
    const float cos_theta_1 = cos(start.x + float(idx_theta + 1u) * delta.x);

    return prob_theta * prob_phi / (delta.y * (cos_theta_0 - cos_theta_1));
}

// ------------------------------------------------------------------------------------------------
// BSDF Measurements
// ------------------------------------------------------------------------------------------------

// Implementation of bsdf_measurement_isvalid() for an MBSDF.
DEVICE bool df_bsdf_measurement_isvalid(Texture_handler_base const* self_base,
                                        unsigned bsdf_measurement_index) {
    Texture_handler const* self =
        static_cast<Texture_handler const*>(self_base);
    return bsdf_measurement_index != 0 &&
        bsdf_measurement_index - 1 < self->num_mbsdfs;
}

// Implementation of df::bsdf_measurement_resolution() function needed by
// generated code, which retrieves the angular and chromatic resolution of the
// given MBSDF. The returned triple consists of: number of equi-spaced steps of
// theta_i and theta_o, number of equi-spaced steps of phi, and number of color
// channels (1 or 3).
DEVICE void df_bsdf_measurement_resolution(
    unsigned result[3], Texture_handler_base const* self_base,
    unsigned bsdf_measurement_index, Mbsdf_part part) {
    Texture_handler const* self =
        static_cast<Texture_handler const*>(self_base);

    if(bsdf_measurement_index == 0 ||
       bsdf_measurement_index - 1 >= self->num_mbsdfs) {
        // invalid MBSDF returns zero
        result[0] = 0;
        result[1] = 0;
        result[2] = 0;
        return;
    }

    Mbsdf const& bm = self->mbsdfs[bsdf_measurement_index - 1];
    const unsigned part_index = static_cast<unsigned>(part);

    // check for the part
    if(bm.has_data[part_index] == 0) {
        result[0] = 0;
        result[1] = 0;
        result[2] = 0;
        return;
    }

    // pass out the information
    result[0] = bm.angular_resolution[part_index].x;
    result[1] = bm.angular_resolution[part_index].y;
    result[2] = bm.num_channels[part_index];
}

INLINEDEVICE float3 bsdf_compute_uvw(const float theta_phi_in[2],
                                     const float theta_phi_out[2]) {
    // assuming each phi is between -pi and pi
    float u = theta_phi_out[1] - theta_phi_in[1];
    if(u < 0.0)
        u += float(2.0 * M_PI);
    if(u > float(1.0 * M_PI))
        u = float(2.0 * M_PI) - u;
    u *= M_ONE_OVER_PI;

    const float v = theta_phi_out[0] * float(2.0 / M_PI);
    const float w = theta_phi_in[0] * float(2.0 / M_PI);

    return make_float3(u, v, w);
}

template <typename T>
INLINEDEVICE T bsdf_measurement_lookup(const hipTextureObject_t& eval_volume,
                                       const float theta_phi_in[2],
                                       const float theta_phi_out[2]) {
    // 3D volume on the GPU (phi_delta x theta_out x theta_in)
    const float3 uvw = bsdf_compute_uvw(theta_phi_in, theta_phi_out);
    return tex3D<T>(eval_volume, uvw.x, uvw.y, uvw.z);
}

// Implementation of df::bsdf_measurement_evaluate() for an MBSDF.
DEVICE void df_bsdf_measurement_evaluate(float result[3],
                                         Texture_handler_base const* self_base,
                                         unsigned bsdf_measurement_index,
                                         float const theta_phi_in[2],
                                         float const theta_phi_out[2],
                                         Mbsdf_part part) {
    Texture_handler const* self =
        static_cast<Texture_handler const*>(self_base);

    if(bsdf_measurement_index == 0 ||
       bsdf_measurement_index - 1 >= self->num_mbsdfs) {
        // invalid MBSDF returns zero
        store_result3(result, 0.0f);
        return;
    }

    const Mbsdf& bm = self->mbsdfs[bsdf_measurement_index - 1];
    const unsigned part_index = static_cast<unsigned>(part);

    // check for the parta
    if(bm.has_data[part_index] == 0) {
        store_result3(result, 0.0f);
        return;
    }

    // handle channels
    if(bm.num_channels[part_index] == 3) {
        const float4 sample = bsdf_measurement_lookup<float4>(
            bm.eval_data[part_index], theta_phi_in, theta_phi_out);
        store_result3(result, sample.x, sample.y, sample.z);
    } else {
        const float sample = bsdf_measurement_lookup<float>(
            bm.eval_data[part_index], theta_phi_in, theta_phi_out);
        store_result3(result, sample);
    }
}

// Implementation of df::bsdf_measurement_sample() for an MBSDF.
DEVICE void
df_bsdf_measurement_sample(float result[3],  // output: theta, phi, pdf
                           Texture_handler_base const* self_base,
                           unsigned bsdf_measurement_index,
                           float const theta_phi_out[2],
                           float const xi[3],  // uniform random values
                           Mbsdf_part part) {
    result[0] = -1.0f;  // negative theta means absorption
    result[1] = -1.0f;
    result[2] = 0.0f;

    Texture_handler const* self =
        static_cast<Texture_handler const*>(self_base);
    if(bsdf_measurement_index == 0 ||
       bsdf_measurement_index - 1 >= self->num_mbsdfs)
        return;  // invalid MBSDFs returns zero

    const Mbsdf& bm = self->mbsdfs[bsdf_measurement_index - 1];
    unsigned part_index = static_cast<unsigned>(part);

    if(bm.has_data[part_index] == 0)
        return;  // check for the part

    // CDF data
    uint2 res = bm.angular_resolution[part_index];
    const float* sample_data = bm.sample_data[part_index];

    // compute the theta_in index (flipping input and output, BSDFs are
    // symmetric)
    unsigned idx_theta_in =
        unsigned(theta_phi_out[0] * M_ONE_OVER_PI * 2.0f * float(res.x));
    idx_theta_in = min(idx_theta_in, res.x - 1);

    // sample theta_out
    //-------------------------------------------
    float xi0 = xi[0];
    const float* cdf_theta = sample_data + idx_theta_in * res.x;
    unsigned idx_theta_out =
        sample_cdf(cdf_theta, res.x, xi0);  // binary search

    float prob_theta = cdf_theta[idx_theta_out];
    if(idx_theta_out > 0) {
        const float tmp = cdf_theta[idx_theta_out - 1];
        prob_theta -= tmp;
        xi0 -= tmp;
    }
    xi0 /= prob_theta;  // rescale for re-usage

    // sample phi_out
    //-------------------------------------------
    float xi1 = xi[1];
    const float* cdf_phi = sample_data + (res.x * res.x) +  // CDF theta block
        (idx_theta_in * res.x + idx_theta_out) * res.y;     // selected CDF phi

    // select which half-circle to choose with probability 0.5
    const bool flip = (xi1 > 0.5f);
    if(flip)
        xi1 = 1.0f - xi1;
    xi1 *= 2.0f;

    unsigned idx_phi_out = sample_cdf(cdf_phi, res.y, xi1);  // binary search
    float prob_phi = cdf_phi[idx_phi_out];
    if(idx_phi_out > 0) {
        const float tmp = cdf_phi[idx_phi_out - 1];
        prob_phi -= tmp;
        xi1 -= tmp;
    }
    xi1 /= prob_phi;  // rescale for re-usage

    // compute theta and phi out
    //-------------------------------------------
    const float2 inv_res = bm.inv_angular_resolution[part_index];

    const float s_theta = float(0.5 * M_PI) * inv_res.x;
    const float s_phi = float(1.0 * M_PI) * inv_res.y;

    const float cos_theta_0 = cosf(float(idx_theta_out) * s_theta);
    const float cos_theta_1 = cosf(float(idx_theta_out + 1u) * s_theta);

    const float cos_theta = cos_theta_0 * (1.0f - xi1) + cos_theta_1 * xi1;
    result[0] = acosf(cos_theta);
    result[1] = (float(idx_phi_out) + xi0) * s_phi;

    if(flip)
        result[1] = float(2.0 * M_PI) - result[1];  // phi \in [0, 2pi]

    // align phi
    result[1] += (theta_phi_out[1] > 0) ?
        theta_phi_out[1] :
        (float(2.0 * M_PI) + theta_phi_out[1]);
    if(result[1] > float(2.0 * M_PI))
        result[1] -= float(2.0 * M_PI);
    if(result[1] > float(1.0 * M_PI))
        result[1] = float(-2.0 * M_PI) + result[1];  // to [-pi, pi]

    // compute pdf
    //-------------------------------------------
    result[2] =
        prob_theta * prob_phi * 0.5f / (s_phi * (cos_theta_0 - cos_theta_1));
}

// Implementation of df::bsdf_measurement_pdf() for an MBSDF.
DEVICE float df_bsdf_measurement_pdf(Texture_handler_base const* self_base,
                                     unsigned bsdf_measurement_index,
                                     float const theta_phi_in[2],
                                     float const theta_phi_out[2],
                                     Mbsdf_part part) {
    Texture_handler const* self =
        static_cast<Texture_handler const*>(self_base);

    if(bsdf_measurement_index == 0 ||
       bsdf_measurement_index - 1 >= self->num_mbsdfs)
        return 0.0f;  // invalid MBSDF returns zero

    const Mbsdf& bm = self->mbsdfs[bsdf_measurement_index - 1];
    unsigned part_index = static_cast<unsigned>(part);

    // check for the part
    if(bm.has_data[part_index] == 0)
        return 0.0f;

    // CDF data and resolution
    const float* sample_data = bm.sample_data[part_index];
    uint2 res = bm.angular_resolution[part_index];

    // compute indices in the CDF data
    float3 uvw = bsdf_compute_uvw(
        theta_phi_in, theta_phi_out);  // phi_delta, theta_out, theta_in
    unsigned idx_theta_in =
        unsigned(theta_phi_in[0] * M_ONE_OVER_PI * 2.0f * float(res.x));
    unsigned idx_theta_out =
        unsigned(theta_phi_out[0] * M_ONE_OVER_PI * 2.0f * float(res.x));
    unsigned idx_phi_out = unsigned(uvw.x * float(res.y));
    idx_theta_in = min(idx_theta_in, res.x - 1);
    idx_theta_out = min(idx_theta_out, res.x - 1);
    idx_phi_out = min(idx_phi_out, res.y - 1);

    // get probability to select theta_out
    const float* cdf_theta = sample_data + idx_theta_in * res.x;
    float prob_theta = cdf_theta[idx_theta_out];
    if(idx_theta_out > 0) {
        const float tmp = cdf_theta[idx_theta_out - 1];
        prob_theta -= tmp;
    }

    // get probability to select phi_out
    const float* cdf_phi = sample_data + (res.x * res.x) +  // CDF theta block
        (idx_theta_in * res.x + idx_theta_out) * res.y;     // selected CDF phi
    float prob_phi = cdf_phi[idx_phi_out];
    if(idx_phi_out > 0) {
        const float tmp = cdf_phi[idx_phi_out - 1];
        prob_phi -= tmp;
    }

    // compute probability to select a position in the sphere patch
    float2 inv_res = bm.inv_angular_resolution[part_index];

    const float s_theta = float(0.5 * M_PI) * inv_res.x;
    const float s_phi = float(1.0 * M_PI) * inv_res.y;

    const float cos_theta_0 = cosf(float(idx_theta_out) * s_theta);
    const float cos_theta_1 = cosf(float(idx_theta_out + 1u) * s_theta);

    return prob_theta * prob_phi * 0.5f / (s_phi * (cos_theta_0 - cos_theta_1));
}

INLINEDEVICE void
df_bsdf_measurement_albedo(float result[2],  // output: max (in case of color)
                                             // albedo for the selected
                                             // direction ([0]) and global ([1])
                           Texture_handler const* self,
                           unsigned bsdf_measurement_index,
                           float const theta_phi[2], Mbsdf_part part) {
    const Mbsdf& bm = self->mbsdfs[bsdf_measurement_index - 1];
    const unsigned part_index = static_cast<unsigned>(part);

    // check for the part
    if(bm.has_data[part_index] == 0)
        return;

    const uint2 res = bm.angular_resolution[part_index];
    unsigned idx_theta =
        unsigned(theta_phi[0] * float(2.0 / M_PI) * float(res.x));
    idx_theta = min(idx_theta, res.x - 1u);
    result[0] = bm.albedo_data[part_index][idx_theta];
    result[1] = bm.max_albedo[part_index];
}

// Implementation of df::bsdf_measurement_albedos() for an MBSDF.
DEVICE void df_bsdf_measurement_albedos(
    float result[4],  // output: [0] albedo refl. for theta_phi
                      //         [1] max albedo refl. global
                      //         [2] albedo trans. for theta_phi
                      //         [3] max albedo trans. global
    Texture_handler_base const* self_base, unsigned bsdf_measurement_index,
    float const theta_phi[2]) {
    result[0] = 0.0f;
    result[1] = 0.0f;
    result[2] = 0.0f;
    result[3] = 0.0f;

    Texture_handler const* self =
        static_cast<Texture_handler const*>(self_base);
    if(bsdf_measurement_index == 0 ||
       bsdf_measurement_index - 1 >= self->num_mbsdfs)
        return;  // invalid MBSDF returns zero

    df_bsdf_measurement_albedo(&result[0], self, bsdf_measurement_index,
                               theta_phi, mi::neuraylib::MBSDF_DATA_REFLECTION);

    df_bsdf_measurement_albedo(&result[2], self, bsdf_measurement_index,
                               theta_phi,
                               mi::neuraylib::MBSDF_DATA_TRANSMISSION);
}

DEVICE void __continuation_callable__sample(Payload* payload, Vec3 dir,
                                            Vec3 hit, Vec3 ng, Vec3 ns,
                                            Vec2 texCoord, float rayTime,
                                            bool front) {
    auto data = getSBTData<DataDesc>();
    SamplerContext& sampler = *payload->sampler;
    MDL::Shading_state_material mat;
    ns = ng;
    mat.normal = v2f(ns);
    mat.geom_normal = v2f(ng);
    mat.position = v2f(hit);
    mat.animation_time = rayTime;
    mat.text_coords = nullptr;  // TODO:texCoords
    // fake tangent
    float3 tu, tv;
    {
        Vec3 bx = { 1.0f, 0.0f, 0.0f }, by = { 0.0f, 1.0f, 0.0f };
        Vec3 t = glm::normalize(fabs(ns.x) < fabs(ns.y) ? glm::cross(ns, bx) :
                                                          glm::cross(ns, by));
        Vec3 bt = glm::cross(t, ns);
        tu = v2f(t), tv = v2f(bt);
    }
    mat.tangent_u = &tu;  // TODO:tangent
    mat.tangent_v = &tv;
    mat.text_results = nullptr;     // TODO:reserve text_results
    mat.ro_data_segment = nullptr;  // TODO:enable_ro_segment
    mat.world_to_object = nullptr;
    mat.object_to_world = nullptr;  // TODO:transform
    mat.object_id = 0;              // TODO:instance
    Texture_handler handler;
    MDL::Resource_data resData;
    resData.shared_data = nullptr;
    resData.texture_handler = &handler;
    bsdf_init(&mat, &resData, nullptr, data->argData);
    Vec3 offset = ng * 0.001f;  // TODO:check offset
    // sample incoming direction
    {
        MDL::Bsdf_sample_data sampleF;
        if(front) {
            sampleF.ior1 = make_float3(1.0f, 1.0f, 1.0f);
            sampleF.ior2.x = MI_NEURAYLIB_BSDF_USE_MATERIAL_IOR;
        } else {
            sampleF.ior1.x = MI_NEURAYLIB_BSDF_USE_MATERIAL_IOR;
            sampleF.ior2 = make_float3(1.0f, 1.0f, 1.0f);
        }

        sampleF.k1 = v2f(-dir);
        sampleF.xi = make_float3(sampler(), sampler(), sampler());
        bsdf_sample(&sampleF, &mat, &resData, nullptr, data->argData);
        payload->f = f2v(sampleF.bsdf_over_pdf);
        payload->wi = f2v(sampleF.k2);
        payload->ori = hit +
            (sampleF.event_type & MDL::BSDF_EVENT_TRANSMISSION ? -offset :
                                                                 offset);
        payload->hit = true;
    }
    // sample light
    {
        MDL::Bsdf_evaluate_data evalF;
        if(front) {
            evalF.ior1 = make_float3(1.0f, 1.0f, 1.0f);
            evalF.ior2.x = MI_NEURAYLIB_BSDF_USE_MATERIAL_IOR;
        } else {
            evalF.ior1.x = MI_NEURAYLIB_BSDF_USE_MATERIAL_IOR;
            evalF.ior2 = make_float3(1.0f, 1.0f, 1.0f);
        }
        LightSample ls =
            sampleOneLight(hit + (front ? offset : -offset), rayTime, sampler);
        evalF.k1 = v2f(-dir);
        evalF.k2 = v2f(ls.wi);
        bsdf_evaluate(&evalF, &mat, &resData, nullptr, data->argData);
        payload->rad = ls.rad * f2v(evalF.bsdf);
        // TODO:light importance sampling
    }
}
